#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

#ifndef block_size_x
    #define block_size_x 128
#endif

#ifndef tile_size_x
    #define tile_size_x 1
#endif

#ifndef use_shared_mem
    #define use_shared_mem 0
#endif


template <int tile_size, int stride, typename T>
__device__ __forceinline__ void fill_shared_mem_tiled_1D(T (&sh_mem)[tile_size*stride], const T *d_mem, int sh_offset, int d_offset, int N) {
    #pragma unroll
    for (int ti=0; ti<tile_size; ti++) {
        if (d_offset+ti*stride < N) {
            sh_mem[sh_offset+ti*stride] = d_mem[d_offset+ti*stride];
        }
    }
}


/*
 * This function performs the main body of work for computing the Gauss transform
 * The parallelization is such that one thread block is created
 * for each item in A, which is of size m. This implies that each thread block
 * does n (size of B) work.
 * The gradient computed in this function is reduced to a single value within the
 * thread block. The same is done for the cross term, which then needs to be
 * reduced in a second kernel. 
 */
template<typename T, int dim>
__device__ __forceinline__ void GaussTransform_blocked_i(const T *A, const T *B,
                const int m, const int n, const T scale_sq, T *d_grad, T *d_cross_term) {

    int tx = threadIdx.x;

    // Specialize BlockReduce for a 1D block of block_size_x threads on type T
    typedef hipcub::BlockReduce<T, block_size_x> BlockReduce;
    // Allocate shared memory for BlockReduce
    __shared__ typename BlockReduce::TempStorage temp_storage;

    T cross_term = 0.0;
    T grad_i[dim];
    for (int d = 0; d < dim; d++) {
        grad_i[d] = 0.0;
    }

    int i = blockIdx.x;

    #if use_shared_mem == 1
    __shared__ T sh_A[dim][block_size_x*tile_size_x];

    #pragma unroll
    for (int d=0; d<dim; d++) {
        fill_shared_mem_tiled_1D<tile_size_x, block_size_x>(sh_A[d], A+d*m, tx, i, m);
    }
    __syncthreads();
    #endif

    //loop parallelized over threads within thread block
    for (int j = tx; j<n; j+=block_size_x) {

        T dist_ij = 0;
        #pragma unroll
        for (int d = 0; d < dim; ++d) {
            dist_ij += (A[i * dim + d] - B[j * dim + d])*(A[i * dim + d] - B[j * dim + d]);
        }
        T cost_ij = exp(-dist_ij/scale_sq);

        #pragma unroll
        for (int d = 0; d < dim; ++d) {
            grad_i[d] -= cost_ij * 2.0 * (A[i * dim + d] - B[j * dim + d]);
        }

        cross_term += cost_ij;
    }

    //reduce grad_i for each d, within the block
    #pragma unroll
    for (int d = 0; d < dim; d++) {
        grad_i[d] = BlockReduce(temp_storage).Sum(grad_i[d]);
        __syncthreads();
    }

    //reduce cross_term within the block, (division by m*n on CPU)
    cross_term = BlockReduce(temp_storage).Sum(cross_term);

    if (tx == 0 && blockIdx.x < m) {
        #pragma unroll
        for (int d = 0; d < dim; d++) {
            d_grad[blockIdx.x * dim + d] = grad_i[d] / (scale_sq * m * n);
        }
        d_cross_term[blockIdx.x] = cross_term;
    }
}


extern "C"
__global__ void
GaussTransform(const double* A, const double* B,
                 int m, int n, double scale_sq, double *grad, double *cross_term) {

    //2-dimensional with double precision
    GaussTransform_blocked_i<double, 2>(A, B, m, n, scale_sq, grad, cross_term);

}

/*
 * Reduce the per thread block cross terms computed in the GaussTransform kernel to single value
 * and divide by (m*n)
 *
 * This kernel is designed to run as single-thread block, because the number of terms to reduce is
 * of size n or m, which is expected to be around 2000 or so. The number of items to reduce
 * is passed as the last argument 'nblocks', which corresponds to the number of thread blocks used
 * by the first kernel.
 */
extern "C"
__global__ void reduce_cross_term(double *output, double *d_cross_term, const int m, const int n, const int nblocks) {

    int tx = threadIdx.x;
    // Specialize BlockReduce for a 1D block of block_size_x threads on type T
    typedef hipcub::BlockReduce<double, block_size_x> BlockReduce;
    // Allocate shared memory for BlockReduce
    __shared__ typename BlockReduce::TempStorage temp_storage;

    double cross_term = 0.0;
    for (int i=tx; i<nblocks; i+=block_size_x) {
        cross_term += d_cross_term[i];
    }

    //reduce to single value within thread block
    cross_term = BlockReduce(temp_storage).Sum(cross_term);

    //thread 0 writes output
    if (tx == 0) {
        output[0] = cross_term / (m*n);
    }

}



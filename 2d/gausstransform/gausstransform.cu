/*
 * Host part for calling GPUGaussTransform from the CPU 
 */
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include <hip/hip_runtime.h>

#include "gausstransform.h"

#ifndef block_size_x
    #define block_size_x 128    //best for GTX 690 and K40
#endif

#include "kernels.cu"



GPUGaussTransform::GPUGaussTransform(int n) {
    //allocate GPU memory for size max_n
    max_n = n;
    dim = 2;
    int elems = max_n * dim;

    hipError_t err;

    //pseudo load balancing across available GPUs
    int count;
    hipGetDeviceCount(&count);
    int id = rand() % count;
    hipSetDevice(id);

    err = hipMalloc((void **)&d_A, elems*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMalloc: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMalloc((void **)&d_B, elems*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMalloc: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMalloc((void **)&d_grad, elems*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMalloc: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMalloc((void **)&d_cross_term, max_n*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMalloc: %s\n", hipGetErrorString(err));
        exit(1);
    }

    err = hipStreamCreate(&stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipStreamCreate: %s\n", hipGetErrorString(err));
        exit(1);
    }

    hipDeviceSynchronize();
} 


GPUGaussTransform::~GPUGaussTransform() {
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_grad);
    hipFree(d_cross_term);
    hipStreamDestroy(stream);
} 

double GPUGaussTransform::compute(const double *A, const double *B,
    int m, int n, double scale, double *grad) {

    double energy;
    hipError_t err;

    //move data to the GPU
    err = hipMemcpyAsync(d_A, A, m*dim*sizeof(double), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyAsync: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMemcpyAsync(d_B, B, n*dim*sizeof(double), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyAsync: %s\n", hipGetErrorString(err));
        exit(1);
    }

    //setup kernel execution parameters
    dim3 threads(block_size_x, 1, 1);
    dim3 grid(m, 1, 1);
    
    //call the first kernel
    double scale_sq = scale * scale;
    GaussTransform<<<grid, threads, 0, stream>>>(d_A, d_B, m, n, scale_sq, d_grad, d_cross_term); 

    //call the second kernel
    dim3 grid2(1, 1, 1);
    reduce_cross_term<<<grid2, threads, 0, stream>>>(d_cross_term, d_cross_term, m, n, m);

    //copy result from GPU memory to host memory
    err = hipMemcpyAsync(grad, d_grad, m*dim*sizeof(double), hipMemcpyDeviceToHost, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyDeviceToHost: %s\n", hipGetErrorString (err));
        exit(1);
    }

    err = hipMemcpyAsync(&energy, d_cross_term, 1*sizeof(double), hipMemcpyDeviceToHost, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyDeviceToHost: %s\n", hipGetErrorString (err));
        exit(1);
    }

    //wait for the GPU stuff to have finished
    hipStreamSynchronize(stream);

    return energy;
}



extern "C"
float test_GaussTransformHost(double *cost, const double* A, const double* B,
            int m, int n, int dim, double scale, double* grad) {

    GPUGaussTransform gpu_gt(1000000);

    *cost = gpu_gt.compute(A, B, m, n, scale, grad);

    return 0.0;
}



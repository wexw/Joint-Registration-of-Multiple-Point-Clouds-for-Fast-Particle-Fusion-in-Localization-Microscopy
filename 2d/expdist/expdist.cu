#include "hip/hip_runtime.h"
/*
 * Host part for calling GPUExpDist from the CPU 
 */

#include <stdint.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdexcept>
#include "expdist.h"

//tuned for Nvidia K40
#ifndef block_size_x //if not using kernel tuner
#define block_size_x 32
#define block_size_y 4
#define tile_size_x 2
#define tile_size_y 4
#define use_shared_mem 1

#endif
#define reduce_block_size 256


#include "kernels.cu"

// Function to select a suitable device
int selectDevice(int requiredMemory) {
    // Get the number of available CUDA devices
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        throw std::runtime_error("No CUDA devices found.");
    }

    for (int i = 0; i < deviceCount; ++i) {
        // Get the properties of the current CUDA device
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        // Check if the device has enough memory for your dataset
        if (deviceProp.totalGlobalMem >= requiredMemory) {
         //   std::cout << "Device " << i << " is available and has enough memory for the dataset." << std::endl;
            hipSetDevice(i);
            return i;
        }

 //else {
            //std::cout << "Device " << i << " does not have enough memory for the dataset." << std::endl;
        //}
    }

    throw std::runtime_error("No device has enough memory for the dataset.");
}




GPUExpDist::GPUExpDist(int n) {
    //allocate GPU memory for size max_n
    max_n = n;
    dim = 2;
    int elems = max_n * dim;
 size_t requiredMemory = 2 * elems * sizeof(double) + 3 * max_n * sizeof(double);
    int deviceID = selectDevice(requiredMemory);
 
    hipError_t err;

    err = hipMalloc((void **)&d_A, elems*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMalloc: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMalloc((void **)&d_B, elems*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMalloc: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMalloc((void **)&d_scale_A, max_n*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMalloc: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMalloc((void **)&d_scale_B, max_n*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMalloc: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMalloc((void **)&d_cross_term, max_n*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMalloc: %s\n", hipGetErrorString(err));
        exit(1);
    }

    err = hipStreamCreate(&stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipStreamCreate: %s\n", hipGetErrorString(err));
        exit(1);
    }

    hipDeviceSynchronize();
} 

GPUExpDist::~GPUExpDist() {
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_scale_A);
    hipFree(d_scale_B);
    hipFree(d_cross_term);
    hipStreamDestroy(stream);
} 

double GPUExpDist::compute(const double *A, const double *B, int m, int n, const double *scale_A, const double *scale_B) {

    double cost;
    hipError_t err;

    //move data to the GPU
    err = hipMemcpyAsync(d_A, A, m*dim*sizeof(double), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyAsync d_A: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMemcpyAsync(d_B, B, n*dim*sizeof(double), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyAsync d_B: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMemcpyAsync(d_scale_A, scale_A, m*sizeof(double), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyAsync d_scale_A: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMemcpyAsync(d_scale_B, scale_B, n*sizeof(double), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyAsync d_scale_B: %s\n", hipGetErrorString(err));
        exit(1);
    }

    //compute number of thread blocks that would be used by the ExpDist kernel for this m and n
    int nblocks = ((int) ceil(m / (block_size_x*tile_size_x)) * (int) ceil(n / (block_size_y*tile_size_y)));

    //setup kernel execution parameters
    dim3 threads(block_size_x, block_size_y, 1);
    dim3 grid(1, 1, 1); //to be overwritten

    //check if the number of thread blocks does not exceed the allocated space
    //if it does, run the ExpDist_column kernel that uses fewer thread blocks
    if (nblocks < max_n) {
        //setup kernel execution parameters
        grid.x = (int) ceilf(m / (float)(block_size_x * tile_size_x));
        grid.y = (int) ceilf(n / (float)(block_size_y * tile_size_y));
    
        //call the first kernel
        ExpDist<<<grid, threads, 0, stream>>>(d_A, d_B, m, n, d_scale_A, d_scale_B, d_cross_term); 

    } else {
        //setup kernel execution parameters
        grid.x = (int) ceilf(m / (float)(block_size_x * tile_size_x));
    
        //call the first kernel
        ExpDist_column<<<grid, threads, 0, stream>>>(d_A, d_B, m, n, d_scale_A, d_scale_B, d_cross_term); 
    }

    //call the second kernel
    dim3 threads2(reduce_block_size, 1, 1);
    dim3 grid2(1, 1, 1);
    reduce_cross_term<<<grid2, threads2, 0, stream>>>(d_cross_term, d_cross_term, m, n, grid.x*grid.y);

    err = hipMemcpyAsync(&cost, d_cross_term, 1*sizeof(double), hipMemcpyDeviceToHost, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyDeviceToHost cross_term: %s\n", hipGetErrorString (err));
        exit(1);
    }

    return cost;
}


extern "C"
float test_GPUExpDistHost(double *cost, const double* A, const double* B,
            int m, int n, int dim, const double *scale_A, const double *scale_B, int max_n) {

    GPUExpDist gpu_expdist(max_n);

    *cost = gpu_expdist.compute(A, B, m, n, scale_A, scale_B);

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error in test_GPUExpDistHost: %s\n", hipGetErrorString (err));
        exit(1);
    }

    return 0.0;
}


